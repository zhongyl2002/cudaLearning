
#include <hip/hip_runtime.h>
#include<stdio.h>

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if(error != hipSuccess){\
        printf("Error: %s:%d, ", __FILE__, __LINE__);\
        printf("code:%d, reason:%s\n", error, hipGetErrorString(error));\
    }\
}

int main(){
    // 检查点
    // CHECK(cudaMemcpy(……))
    return 0;
}