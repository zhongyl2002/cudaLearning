#include<hip/hip_runtime.h>
#include<stdio.h>

__global__ void checkIndex(void){
    printf("threadIdx:(%d, %d, %d) blockIdx:(%d %d %d) blockDim:(%d %d %d) gridDim:(%d %d %d)\n",
            threadIdx.x, threadIdx.y, threadIdx.z,
            blockIdx.x, blockIdx.y, blockIdx.z,
            blockDim.x, blockDim.y, blockDim.z,
            gridDim.x, gridDim.y, gridDim.z);
}

int main(){
    int nElem = 6;

    dim3 block(3);
    dim3 grid ((nElem + block.x - 1) / block.x);

    printf("gird : %d %d %d\n", grid.x, grid.y, grid.z);
    printf("block : %d %d %d\n", block.x, block.y, block.z);

    checkIndex <<<grid, block>>> ();

    hipDeviceReset();

    return 0;
}