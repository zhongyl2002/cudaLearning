#include<hip/hip_runtime.h>
#include<stdio.h>

int main(){
    int nElem = 1024;

    dim3 block(1024);
    // nElem + block.x - 1应该是为了向上取整
    dim3 grid((nElem + block.x - 1) / block.x);

    block.x = 512;
    grid.x = (nElem + block.x - 1) / block.x;
    printf("gird.x = %d, block.x = %d\n", grid.x, block.x);

    block.x = 256;
    grid.x = (nElem + block.x - 1) / block.x;
    printf("gird.x = %d, block.x = %d\n", grid.x, block.x);

    block.x = 128;
    grid.x = (nElem + block.x - 1) / block.x;
    printf("gird.x = %d, block.x = %d\n", grid.x, block.x);

    hipDeviceReset();
    return 0;

}